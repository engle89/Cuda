#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <hip/hip_vector_types.h>
#include ""
#include "cutil_math.h" 

#define M_PI 3.14159265359f  
#define width 512  
#define height 384 
#define samples 4096

struct Ray {
	float3 origin; 
	float3 direction;  
	__device__ Ray(float3 o, float3 d) : origin(o), direction(d) {}
};

enum Material { Diffuse, Specular, Refraction }; 

struct Sphere {

	float radius;            
	float3 position, emission, color; 
	Material material;          

	__device__ float intersect_sphere(const Ray &r) const {

		float3 op = r.origin - position;
		float t, epsilon = 0.0001f;
		float b = dot(op, r.direction);

		float discriminant = b * b - dot(op, op) + radius * radius;
		if (discriminant < 0)
			return 0;
		else
			discriminant = sqrtf(discriminant);

		t = -b - discriminant;
		if (t > epsilon)
			return t;
		else
		{
			t = -b + discriminant;
			if (t > epsilon)
				return t;
			else
				return 0;
		}
	}
};

// Scene
__constant__ Sphere spheres[] = {
{ 1e5f,{ 1e5f + 1.0f, 40.8f, 81.6f },{ 0.0f, 0.0f, 0.0f },{ 0.75f, 0.25f, 0.25f }, Diffuse }, //Left 
{ 1e5f,{ -1e5f + 99.0f, 40.8f, 81.6f },{ 0.0f, 0.0f, 0.0f },{ .25f, .25f, .75f }, Diffuse }, //Rght 
{ 1e5f,{ 50.0f, 40.8f, 1e5f },{ 0.0f, 0.0f, 0.0f },{ .75f, .75f, .75f }, Diffuse }, //Back 
{ 1e5f,{ 50.0f, 40.8f, -1e5f + 600.0f },{ 0.0f, 0.0f, 0.0f },{ 1.00f, 1.00f, 1.00f }, Diffuse }, //Frnt 
{ 1e5f,{ 50.0f, 1e5f, 81.6f },{ 0.0f, 0.0f, 0.0f },{ .75f, .75f, .75f }, Diffuse }, //Botm 
{ 1e5f,{ 50.0f, -1e5f + 81.6f, 81.6f },{ 0.0f, 0.0f, 0.0f },{ .75f, .75f, .75f }, Diffuse }, //Top 
{ 16.5f,{ 27.0f, 16.5f, 47.0f },{ 0.0f, 0.0f, 0.0f },{ 1.0f, 1.0f, 1.0f }, Specular}, // small sphere 1
{ 16.5f,{ 73.0f, 16.5f, 78.0f },{ 0.0f, 0.0f, 0.0f },{ 1.0f, 1.0f, 0.5f }, Diffuse }, // small sphere 2
{ 16.5f,{ 50.0f, 50.0f, 50.0f}, { 0.0f, 0.0f, 0.0f },{ 1.0f, 1.0f, 1.0f }, Refraction }, //small sphere 3
{ 600.0f,{ 50.0f, 681.6f - .77f, 81.6f },{ 2.0f, 1.8f, 1.6f },{ 0.0f, 0.0f, 0.0f }, Diffuse }  // Light
};

__device__ inline bool intersect_scene(const Ray &r, float &t, int &id) {

	float n = sizeof(spheres) / sizeof(Sphere), d, inf = t = 1e20;  
	for (int i = int(n); i--;) 
		if ((d = spheres[i].intersect_sphere(r)) && d < t) {  
			t = d;  
			id = i; 
		}
	return t < inf;
}

__device__ static float getrandom(unsigned int *seed0, unsigned int *seed1) {
	*seed0 = 36969 * ((*seed0) & 65535) + ((*seed0) >> 16); 
	*seed1 = 18000 * ((*seed1) & 65535) + ((*seed1) >> 16);

	unsigned int ires = ((*seed0) << 16) + (*seed1);

	union {
		float f;
		unsigned int ui;
	} res;

	res.ui = (ires & 0x007fffff) | 0x40000000;  

	return (res.f - 2.f) / 2.f;
}

__device__ float3 radiance(Ray &r, unsigned int *s1, unsigned int *s2) { 
	float3 accucolor = make_float3(0.0f, 0.0f, 0.0f); 
	float3 mask = make_float3(1.0f, 1.0f, 1.0f);

	// ray bounce loop no recursionin device 
	for (int bounces = 0; bounces < 4; bounces++) { 

		float t;         
		int id = 0;        

		//miss
		if (!intersect_scene(r, t, id))
			return make_float3(0.0f, 0.0f, 0.0f); 

		const Sphere &obj = spheres[id];  
		float3 x = r.origin + r.direction*t;          
		float3 n = normalize(x - obj.position);    
		float3 nl = dot(n, r.direction) < 0 ? n : n * -1; 
		
		//emissive
		accucolor += mask * obj.emission;

		//diffuse
		if (obj.material == Diffuse)
		{
			float r1 = 2 * M_PI * getrandom(s1, s2);
			float r2 = getrandom(s1, s2);
			float r2s = sqrtf(r2);

			float3 w = nl;
			float3 u = normalize(cross((fabs(w.x) > .1 ? make_float3(0, 1, 0) : make_float3(1, 0, 0)), w));
			float3 v = cross(w, u);

			float3 d = normalize(u*cos(r1)*r2s + v * sin(r1)*r2s + w * sqrtf(1 - r2));

			r.origin = x + nl * 0.05f; //offset for self intersection
			r.direction = d;

			mask *= obj.color;
			mask *= dot(d, nl);  // weigh light contribution using cosine of angle between incident light and normal
			mask *= 2;          // fudge factor
		}
		//specular
		else if (obj.material == Specular)
		{
			r.origin = x + nl * 0.07f;
			r.direction = r.direction - n * 2 * dot(n, r.direction);

			mask *= obj.color;
			mask *= dot(r.direction, nl);
			mask *= 2;
		}
		//refraction
		else 
		{
			r.origin = x + nl * 0.05f;
			r.direction = r.direction - n * 2 * dot(n, r.direction);

			
			bool into = (dot(n, nl) > 0);
			double nc = 1;
			double nt = 1.5; //IOR for glass is 1.5
			double nnt = into ? nc / nt : nt / nc;
			double ddn = dot(r.direction, nl);
			double cos2t;

			//total internal reflection
			if ((cos2t = 1 - nnt * nnt*(1 - ddn * ddn)) < 0)
			{
				mask *= obj.color;
				mask *= dot(r.direction, nl);
				mask *= 2;
			}
			//otherwise, choose refraction
			else
			{
				r.direction = normalize((r.direction*nnt - n * ((into ? 1 : -1)*(ddn*nnt + sqrt(cos2t)))));
				double a = nt - nc, b = nt + nc, R0 = a * a / (b*b), c = 1 - (into ? -ddn : dot(r.direction, n));
				double Re = R0 + (1 - R0)*c*c*c*c*c;
				double Tr = 1 - Re;
				double P = 0.25 + 0.5*Re;
				double RP = Re / P;
				double TP = Tr / (1 - P);
				mask *= TP;
				mask *= obj.color;
				mask *= dot(r.direction, nl);
				mask *= 2;
			}		
			
		}
	}

	return accucolor;
}

__global__ void render_kernel(float3 *output) {

	unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
	unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;

	unsigned int i = (height - y - 1)*width + x; 

	unsigned int s1 = x; 
	unsigned int s2 = y;

	Ray cam(make_float3(50, 52, 295.6), normalize(make_float3(0, -0.042612, -1))); 
	float3 cx = make_float3(width * .5135 / height, 0.0f, 0.0f); 
	float3 cy = normalize(cross(cx, cam.direction)) * .5135;
	float3 r;      

	r = make_float3(0.0f); 

	for (int s = 0; s < samples; s++) {  

		float3 d = cam.direction + cx * ((.25 + x) / width - .5) + cy * ((.25 + y) / height - .5);

		r = r + radiance(Ray(cam.origin + d * 40, normalize(d)), &s1, &s2)*(1. / samples);
	}       

	output[i] = make_float3(clamp(r.x, 0.0f, 1.0f), clamp(r.y, 0.0f, 1.0f), clamp(r.z, 0.0f, 1.0f));
}

inline float clamp(float x) { return x < 0.0f ? 0.0f : x > 1.0f ? 1.0f : x; }

inline int toInt(float x) { return int(pow(clamp(x), 1 / 2.2) * 255 + .5); } 

int main() {

	float3* output_h = new float3[width*height]; 
	float3* output_d;    

	hipMalloc(&output_d, width * height * sizeof(float3));

	dim3 block(8, 8, 1);
	dim3 grid(width / block.x, height / block.y, 1);

	printf("CUDA initialised.\nStart rendering...\n");

	render_kernel << < grid, block >> >(output_d);

	hipMemcpy(output_h, output_d, width * height * sizeof(float3), hipMemcpyDeviceToHost);

	hipFree(output_d);

	printf("Done!\n");

	FILE *f = fopen("smallptcuda.ppm", "w");
	fprintf(f, "P3\n%d %d\n%d\n", width, height, 255);
	for (int i = 0; i < width*height; i++)  
		fprintf(f, "%d %d %d ", toInt(output_h[i].x),
			toInt(output_h[i].y),
			toInt(output_h[i].z));

	printf("Saved image to 'smallptcuda.ppm'\n");

	delete[] output_h;
	system("PAUSE");
}